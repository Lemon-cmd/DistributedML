#include "hip/hip_runtime.h"
#include "modules.h"
#include "cu_mat.h"
#include "dense.h"
#include "clockcycle.h"

int main()
{
	/*
	Matrix X(3, 1), Y(3, 1);

	X.Random();
	Y.Constant(1);

	X.ToHost();
	Y.ToHost();

	std::cout << X << '\n';
	std::cout << Y << '\n';

	X = X.log();
	Y = Y.log();

	X.ToHost();
	Y.ToHost();

	std::cout << X << '\n';
	std::cout << Y << '\n';

	Matrix A = X.T().dot(Y);
	A.ToHost();
	std::cout << A << '\n';
	*/

	float accuracy = 0.0;

	Matrix X(100, 5), Y(100, 10);
	X.Random();
	Y.Constant(1);
	X.ToDevice();
	Y.ToDevice();

	std::vector<Dense> network(3);

	network[0] = Dense(100, "identity");
	network[0].init(5);
	network[0].ToDevice();

	for (uint j = 1; j < network.size(); j++)
	{
		network[j] = Dense(100 - j, "identity");
		network[j].init(network[j - 1].OutShape());
		network[j].ToDevice();
	}

	network[0].forward(X);
	for (uint j = 1; j < network.size(); j++)
	{
		network[j].forward(network[j - 1].Get_H());
	}

	float loss = network.back().MSELoss(Y, accuracy);
	std::cout << "L: " << loss << std::endl;

	return 0;
}
