#include "hip/hip_runtime.h"
#include "modules.h"
#include "dense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void gpu_blas_mmul(const float *A, const float *B, float *C,
			   const int m, const int k, const int n, hipblasHandle_t& handle)
{

	int lda = m, ldb = k, ldc = m;
	const float alpha = 1, beta = 0;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				m, n, k,
				&alpha, A, lda,
				B, ldb, &beta,
				C, ldc);
}

__global__ void add_arrs(const float *A, const float *B, float *C, size_t size)
{
	const uint stride = blockDim.x * gridDim.x;
    const uint tid = blockDim.x * blockIdx.x + threadIdx.x;
	
	for (uint j = tid; j < size; j += stride) 
	{
		C[j] = A[j] + B[j];
	}
}



int main()
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);


	Eigen::MatrixXf X = Eigen::MatrixXf::Random(2, 2), Y = Eigen::MatrixXf::Random(2, 1);

	std::cout << X << "\n\n" << Y << std::endl;

	float *x, *y, *r, *o;

	hipMalloc(&x, X.size() * sizeof(float));
	hipMalloc(&y, Y.size() * sizeof(float));
	
	hipMemcpy(x, X.data(), X.size() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y, Y.data(), Y.size() * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&r, X.rows() * Y.cols() * sizeof(float));

	//add_arrs <<<1, 1>>> (x, y, r, X.size());
	//hipDeviceSynchronize();

	gpu_blas_mmul(x, y, r, X.rows(), X.cols(), Y.cols(), handle);
	hipDeviceSynchronize();
	
	o = (float*) calloc(X.rows() * Y.cols(), sizeof(float));
	hipMemcpy(o, r, X.rows() * Y.cols() * sizeof(float), hipMemcpyDeviceToHost);

	Eigen::MatrixXf R = Eigen::Map <Eigen::MatrixXf> (o, X.rows(), Y.cols());
	std::cout << '\n' <<  R << '\n';

	hipFree(x);
	hipFree(y);
	hipFree(r);
	free(o);
	hipblasDestroy(handle);	
}
