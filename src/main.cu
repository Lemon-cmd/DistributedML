#include "hip/hip_runtime.h"
#include "modules.h"
#include "cu_mat.h"
#include "dense.h"
#include "clockcycle.h"
#include "parse_mnist.h"

int main()
{
	/*
	Matrix X(3, 1), Y(3, 1);

	X.Constant(1);
	Y.Constant(1);

	X.ToHost();
	Y.ToHost();

	std::cout << X << "\n\n"
			  << Y << std::endl;

	std::cout << X.compare(Y) << std::endl;
	*/

	std::vector<Matrix> train_images, train_labels;
	load_mnist("../data/train-images-idx3-ubyte",
			   "../data/train-labels-idx1-ubyte",
			   100, train_images, train_labels);

	std::vector<Matrix> test_images, test_labels;
	load_mnist("../data/t10k-images-idx3-ubyte",
			   "../data/t10k-labels-idx1-ubyte",
			   100, test_images, test_labels);

	std::cout << train_images.size() << '\n'
			  << train_labels.size() << '\n';

	std::vector<Dense> network(3);

	std::cout << "Init:\n";

	network[0] = Dense(30, "relu", 0.01);
	network[0].init(28 * 28);

	for (uint j = 1; j < network.size() - 1; j++)
	{
		network[j] = Dense(50, "tanh", 0.1);
		network[j].init(network[j - 1].OutShape());
	}

	network.back() = Dense(1, "sigmoid", 0.001);
	network.back().init(network[network.size() - 2].OutShape());

	std::cout << "Training:\n";
	uint epochs = 10;

	for (uint e = 0; e < epochs; e++)
	{
		float loss = 0.0, acc = 0.0;

		for (uint k = 0; k < train_images.size(); k++)
		{
			float acc_batch = 0.0;
			// Forward pass
			network[0].forward(train_images[k]);
			for (uint j = 1; j < network.size(); j++)
			{
				network[j].forward(network[j - 1].Get_H());
			}

			loss += network.back().BCELoss(train_labels[k], acc_batch);
			acc += acc_batch;

			// network.back().ToHost();
			// std::cout << "H: " << network.back().Get_H() << std::endl;

			// Update
			network.back().update();

			for (int j = network.size() - 2; j >= 0; j--)
			{
				network[j].set_delta(network[j + 1].Get_delta());
				network[j].update();
			}
		}

		loss /= train_images.size();
		acc /= train_images.size();

		std::cout << "L: " << loss << " A: " << acc << std::endl;
	}
}
